#include "hip/hip_runtime.h"
#include <iostream>
#include <iterator>
#include <fstream>
#include <cmath>
#include <cstdint>
#include <chrono>
#include <vector>
#include <string>

#include "../srcVec3f/Consts.cuh"
#include "../srcVec3f/cuda_util.cuh"
#include "../srcVec3f/Vec3f.cuh"
#include "../srcVec3f/Light.cuh"
#include "../srcVec3f/Sphere.cuh"
#include "../srcVec3f/Ray.cuh"

// CPU Timer
auto start_CPU = std::chrono::high_resolution_clock::now();

void start_CPU_timer(){
    start_CPU = std::chrono::high_resolution_clock::now();
}

long stop_CPU_timer(const char* info){
    auto elapsed = std::chrono::high_resolution_clock::now() - start_CPU;
    long microseconds = std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
    std::cout << microseconds << " microseconds\t\t" << info << std::endl;
    return microseconds;
}

// Maximum of two floats
__device__ constexpr float f_max(float a, float b) {
  return a > b ? a : b;
}

// Convert vector with normalized values to color
__device__ Color convert_to_color(const Vec3f& v) {
  return Color(static_cast<int>(1 * ((v.x()) * 255.999)), static_cast<int>(1 * ((v.y()) * 255.999)), static_cast<int>(1 * ((v.z()) * 255.999)));
}

// Find the closest intersecting sphere of a ray if it exists and set the closest intersection of all spheres if they exist
__device__ int get_closest_intersection(Sphere* spheres, const Ray& r, float* intersections) {
  int hp = -1;
  
  // Find all the spheres which the ray intersects with
  for (int ii = 0; ii < OBJ_COUNT; ii++) {
    intersections[ii] = r.has_intersection(spheres[ii]);
  }

  // If there is only one sphere in the scene
  if (OBJ_COUNT == 1) {
    // No found intersections
    if (intersections[0] < 0) {
      hp = -1;
    }
    // Found intersection
    else {
      hp = 0;
    }
  }
  // Multiple spheres in the scene
  else if (OBJ_COUNT > 1) {
    float min_val = 100.0; // Current shortest distance to intersection

    for (int ii = 0; ii < OBJ_COUNT; ii++) {
      // Skip as intersection was behind the ray or did not exist
      if (intersections[ii] < 0.0) {
        continue;
      }
      // Current intersection is closer than the previous one
      else if (intersections[ii] < min_val) {
          min_val = intersections[ii];
          hp = ii;
      }
    }
  }

  return hp;
}

// Calculate the color to display at the intersection between ray and sphere
__device__ Color get_color_at(const Ray &r, float intersection, Light* light, const Sphere &sphere, Sphere* spheres, Vec3f* origin) {
  float shadow = 1; // Initialize shadow to full brightness

  Vec3f normal = sphere.get_normal_at(r.at(intersection));

  // Normalized vector from intersection point to camera
  Vec3f to_camera(*origin - r.at(intersection));
  to_camera = to_camera.normalize();

  // Normalized vector from intersection point to light source
  Vec3f light_ray(light->get_position() - r.at(intersection));
  light_ray = light_ray.normalize();

  // Normalized vector of the reflected ray when hitting a sphere
  Vec3f reflection_ray = (-1 * light_ray) - 2 * dot((-1 * light_ray), normal) * normal;
  reflection_ray = reflection_ray.normalize();

  // Reflection ray from intersection point
  Ray rr(r.at(intersection) + 0.001 * normal, reflection_ray); // Offset ray from surface so that it does not hit the same surface it just reflected away from
  float intersections[OBJ_COUNT];
  int hp = get_closest_intersection(spheres, rr, intersections); // What the reflection ray hits
  float reflect_shadow = 1;
  Color reflect_color = Vec3f(BGD_R, BGD_G, BGD_B) / 255; // Set color in reflection to be background color by default

  // Reflection ray hit a sphere
  if (hp != -1) {
    // Ray from intersection point on the sphere that is reflected towards the light source
    Ray rs(rr.at(intersections[hp]) + 0.001 * spheres[hp].get_normal_at(rr.at(intersections[hp])), light->get_position() - rr.at(intersections[hp]) + 0.001 * spheres[hp].get_normal_at(rr.at(intersections[hp])));

    // Check if ray from intersection point on the sphere that is reflected towards the light source hits any sphere that creates a shadow
    for (int i = 0; i < OBJ_COUNT; ++i) {
      // There is a a sphere creating a shadow on the reflected sphere
      if (rs.has_intersection(spheres[i]) > 0.000001f) {
        reflect_shadow = 0.35;
      }
    }

    reflect_color = reflect_shadow * spheres[hp].color;
  }

  // Calculate ambient, diffuse, and specular components of the light
  Vec3f ambient = light->get_ambient() * light->get_color(); 
  Vec3f diffuse = (light->get_diffuse() * f_max(dot(light_ray, normal), 0.0f)) * light->get_color();
  Vec3f specular = light->get_specular() * pow(f_max(dot(reflection_ray, to_camera), 0.0f), 32) * light->get_color();
  
  // Ray from interesection point on original sphere towards the light source
  Ray shadow_ray(r.at(intersection) + (0.001f * normal), light->get_position() - (r.at(intersection) + 0.001f * normal));

  // Check if ray from intersection point on original sphere towards the light source hits any sphere that creates a shadow
  for (int i = 0; i < OBJ_COUNT; ++i) {
    // There is a sphere creating a shadow on the original sphere
    if (shadow_ray.has_intersection(spheres[i]) > 0.000001f) {
      shadow = 0.35;
    }
  }

  // Final color before adding the shadow on the original sphere
  Vec3f all_light = (ambient + diffuse + specular).cap(1) & (0.55 * (sphere.color - reflect_color) + reflect_color).cap(1);
  
  return convert_to_color(shadow * all_light);
}

// Cast one ray per pixel
__global__ void cast_ray(Vec3f* fb, Sphere* spheres, Light* light, Vec3f* origin) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  int j = (blockIdx.y * blockDim.y) + threadIdx.y;

  int tid = (j*WIDTH) + i;

  // Outside rendered pixels
  if (i >= WIDTH || j >= HEIGHT) {
    return;
  }

  // Calculate the ray from the position of the camera toward the 3D scene through the current pixel on the 2D image plane
  Vec3f ij(2 * (float((i) + 0.5) / (WIDTH - 1)) - 1, 1 - 2 * (float((j) + 0.5) / (HEIGHT - 1)), -1); // Direction vector for the ray
  Vec3f dir(ij - *origin);
  Ray r(*origin, dir);

  float intersections[OBJ_COUNT]; // The closest intersections of each sphere
  int hp = get_closest_intersection(spheres, r, intersections); // The closest intersecting sphere

  // Did not hit any spheres (background color)
  if (hp == -1) {
    fb[tid] = Vec3f(BGD_R, BGD_G, BGD_B);
  }
  // Did hit a sphere
  else {
    Color color = get_color_at(r, intersections[hp], light, spheres[hp], spheres, origin);
    fb[tid] = color;
  }
}

void initDevice(int& device_handle) {
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);
  printDeviceProps(devProp);

  hipSetDevice(device_handle);
}

void run_kernel(const int pixels, Vec3f* fb, Sphere* spheres, Light* light, Vec3f* origin) {
  // Device
  Vec3f* fb_device = nullptr;
  Sphere* spheres_dv = nullptr;
  Light* light_dv = nullptr;
  Vec3f* origin_dv = nullptr;

  start_CPU_timer();

  // Device memory allocation
  checkErrorsCuda(hipMalloc((void**) &fb_device, sizeof(Vec3f) * pixels));
  checkErrorsCuda(hipMalloc((void**) &spheres_dv, sizeof(Sphere) * OBJ_COUNT));
  checkErrorsCuda(hipMalloc((void**) &light_dv, sizeof(Light) * 1));
  checkErrorsCuda(hipMalloc((void**) &origin_dv, sizeof(Vec3f) * 1));

  stop_CPU_timer("Device memory allocation");
  start_CPU_timer();

  // Host to device memory transfer
  checkErrorsCuda(hipMemcpy((void*) fb_device, fb, sizeof(Vec3f) * pixels, hipMemcpyHostToDevice));
  checkErrorsCuda(hipMemcpy((void*) spheres_dv, spheres, sizeof(Sphere) * OBJ_COUNT, hipMemcpyHostToDevice));
  checkErrorsCuda(hipMemcpy((void*) light_dv, light, sizeof(Light) * 1, hipMemcpyHostToDevice));
  checkErrorsCuda(hipMemcpy((void*) origin_dv, origin, sizeof(Vec3f) * 1, hipMemcpyHostToDevice));

  stop_CPU_timer("HtoD memory transfer");
  start_CPU_timer();

  // Launch kernel
  dim3 blocks(WIDTH / TPB, HEIGHT / TPB);
  cast_ray<<<blocks, dim3(TPB, TPB)>>>(fb_device, spheres_dv, light_dv, origin_dv);

  hipDeviceSynchronize();

  stop_CPU_timer("CUDA kernel");
  start_CPU_timer();

  // Device to host memory transfer
  checkErrorsCuda(hipMemcpy(fb, fb_device, sizeof(Vec3f) * pixels, hipMemcpyDeviceToHost));

  stop_CPU_timer("DtoH memory transfer");
  start_CPU_timer();

  // Free device memory
  checkErrorsCuda(hipFree(fb_device));
  checkErrorsCuda(hipFree(spheres_dv));
  checkErrorsCuda(hipFree(light_dv));
  checkErrorsCuda(hipFree(origin_dv));

  stop_CPU_timer("Freeing device memory");
}

int main(int argc, char *argv[]) {
  int write_to_file = true;
  
  if (argc == 2) {
    write_to_file = atoi(argv[1]);
  }

  std::ofstream file("img.ppm");

  const int pixels = WIDTH * HEIGHT;
  int device_handle = 0;

  int deviceCount = 0;
  checkErrorsCuda(hipGetDeviceCount(&deviceCount));

  if (deviceCount == 0) {
    std::cerr << "initDevice(): No CUDA Device found." << std::endl;
    return EXIT_FAILURE;
  }

  initDevice(device_handle);
  
  std::cout << "===========================================" << std::endl;

  // Host memory allocation
  start_CPU_timer();

  // Frame buffer for all pixels
  Vec3f* frame_buffer = new Vec3f[pixels];
  std::vector<std::string> mem_buffer;

  // Create an array of spheres
  Sphere *spheres = new Sphere[OBJ_COUNT] {
    Sphere(1000, Vec3f(0, -1002, 0), Color(0.5, 0.5, 0.5)),
    Sphere(0.25, Vec3f(-1.5, -0.25, -4), Color(1.0, 0.0, 0.0)),
    Sphere(0.25, Vec3f(-1.0, -0.25, -4), Color(1.0, 0.5, 0.0)),
    Sphere(0.25, Vec3f(-0.5, -0.25, -4), Color(1.0, 1.0, 0.0)),
    Sphere(0.25, Vec3f(0, -0.25, -4), Color(0.0, 1.0, 0.0)),
    Sphere(0.25, Vec3f(0.5, -0.25, -4), Color(0.0, 1.0, 1.0)),
    Sphere(0.25, Vec3f(1.0, -0.25, -4), Color(0.0, 0.0, 1.0)),

    Sphere(0.25, Vec3f(1.5, -0.25, -4), Color(0.5, 0.0, 1.0)),
    Sphere(0.25, Vec3f(-1.25, 0.25, -3), Color(1.0, 0.0, 0.5)),
    Sphere(0.25, Vec3f(-0.75, 0.25, -3), Color(0.5, 0.0, 0.5)),
    Sphere(0.25, Vec3f(-0.25, 0.25, -3), Color(0.5, 0.5, 0.5)),
    Sphere(0.25, Vec3f(0.25, 0.25, -3), Color(1.0, 1.0, 0.5)),
    Sphere(0.25, Vec3f(0.75, 0.25, -3), Color(0.0, 1.0, 0.5)),

    Sphere(0.25, Vec3f(1.25, 0.25, -3), Color(0.0, 0.5, 1.0)),
    Sphere(0.25, Vec3f(-1.0, 0.75, -2), Color(1.0, 0.5, 0.0)),
    Sphere(0.25, Vec3f(-0.5, 0.75, -2), Color(0.0, 1.0, 1.0)),
    Sphere(0.25, Vec3f(0, 0.75, -2), Color(0.5, 0.0, 1.0)),
    Sphere(0.25, Vec3f(0.5, 0.75, -2), Color(0.0, 0.5, 0.0)),
    Sphere(0.25, Vec3f(1.0, 0.75, -2), Color(1.0, 1.0, 1.0)),
  };

  // Origin of the camera
  Vec3f *origin = new Vec3f(0, 0, 1);

  // Light source in the scene
  Light *light = new Light(Vec3f(1, 1, 1), Vec3f(1, 1, 1));
  // light->set_light(.2, .5, .5);
  light->set_light(.1, .7, .7);

  stop_CPU_timer("Host memory allocation");

  std::cout << ">> Starting kernel for " << WIDTH << "x" << HEIGHT << " image..." << std::endl;
  run_kernel(pixels, frame_buffer, spheres, light, origin);
  std::cout << ">> Finished kernel" << std::endl;

  std::cout << ">> Saving Image..." << std::endl;

  start_CPU_timer();

  if (write_to_file == 1) {
    // Write from the frame buffer to image file
    file << "P3" << "\n" << WIDTH << " " << HEIGHT << "\n" << "255\n";

    for (std::size_t i = 0; i < pixels; ++i) {
      mem_buffer.push_back(std::to_string((int) frame_buffer[i].x()) + " " + std::to_string((int) frame_buffer[i].y()) + " " + std::to_string((int) frame_buffer[i].z()));
    }

    std::ostream_iterator<std::string> output_iterator(file, "\n");
    std::copy(mem_buffer.begin(), mem_buffer.end(), output_iterator);
  }

  stop_CPU_timer("Writing to file");

  start_CPU_timer();

  // Free host memory
  delete[] frame_buffer;
  delete origin;
  delete light;
  delete[] spheres;

  stop_CPU_timer("Freeing host memory");

  std::cout << "===========================================" << std::endl;

  return EXIT_SUCCESS;
}
